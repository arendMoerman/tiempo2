#include "hip/hip_runtime.h"
#include "InterfaceCUDA.h"

/*! \file Kernels.cu
    \brief Definitions of CUDA kernels for TiEMPO2.

    author: Arend Moerman
*/

__constant__ float const_effs[CEFFSSIZE];   // Contains constant efficiencies:chain, gnd, mir, pb 
__constant__ float cPI;                     // Pi
__constant__ float cCL;                     // speed of light
__constant__ float cHP;                     // Planck constant
__constant__ float cKB;                     // Boltzmann constant
__constant__ float cdt;                     // Timestep

__constant__ float cfreq_chop;              // Chopping frequency
__constant__ float cfreq_nod;               // Nodding frequency
__constant__ float cdAz_chop;               // Chopping throw
__constant__ float ct0;                     // Starting time

__constant__ float ch_column;               // Column height
__constant__ float cv_wind;                 // Windspeed
__constant__ int cnx;                       // Number of x points in screen
__constant__ int cny;                       // Number of y points in screen

__constant__ float cdelta;                  // Bandgap energy of MKID
__constant__ float cfreq_sample;            // Sampling frequency of readout

__constant__ int cnt;                       // Number of time evals
__constant__ int cnf_filt;                  // Number of filter freqs
__constant__ int cnf_src;                   // Number of source frequencies
__constant__ int cnf_atm;                   // Number of atmosphere frequencies
__constant__ int cnPWV_atm;                 // Number of atmosphere PWV values

__constant__ int cnAz;                      // Number of az points per freq slice
__constant__ int cnEl;                      // Number of el points per freq slice

__constant__ int cOFF_empty;                // Interpolate on source or no source in OFF position

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }

/**
  Check CUDA API error status of call.
 
  Wrapper for finding errors in CUDA API calls.
 
  @param code The errorcode returned from failed API call.
  @param file The file in which failure occured.
  @param line The line in file in which error occured.
  @param abort Exit code upon error.
 */
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

/**
  Calculate Planckian distribution.

  Used for calculating blackbody intensities of atmosphere, ground and telescope.

  @param T Temperature of blackbody, in Kelvin.
  @param nu Frequency at which to evaluate blackbody, in Hertz.

  @returns Blackbody intensity.
 */
__host__ float getPlanck(float T, float nu)
{
    float CL = 2.9979246e8; // m s^-1
    float HP = 6.62607015e-34;
    float KB = 1.380649e-23;
    
    float prefac = 2 * HP * nu*nu*nu / (CL*CL);
    float dist = 1 / (exp(HP*nu / (KB*T)) - 1);
    
    return prefac * dist;
}

/**
  Calculate sign of number.

  Used in determining chop-nod state.

  @param val Value of number.
 */
__device__ __inline__ void sgn(float val, int &out) {
    out = (float(0) < val) - (val < float(0));
}

/**
  Initialize CUDA.
 
  Instantiate program and populate constant memory.
 
  @param instrument CuInstrument object containing instrument to be simulated.
  @param telescope CuTelescope object containing telescope to be simulated.
  @param simparams CuSimParams object containing simulation parameters.
  @param source CuSource object containing source definitions.
  @param atmosphere CuAtmosphere object containing atmosphere parameters.
  @param nThreads Number of CUDA threads per block.
 
  @return BT Array of two dim3 objects, containing number of blocks per grid and number of threads per block.
 */
 __host__ std::array<dim3, 2> initCUDA(CuInstrument *instrument, CuTelescope *telescope, CuSimParams *simparams, CuSource *source, CuAtmosphere *atmosphere, int nThreads)
 {
     int nBlocks = ceil(simparams->nTimes / nThreads);

     // Calculate nr of blocks per grid and nr of threads per block
     dim3 nrb(nBlocks); dim3 nrt(nThreads);

     float PI = 3.1415926; /* pi */
     float CL = 2.9979246e8; // m s^-1
     float HP = 6.62607015e-34;
     float KB = 1.380649e-23;

     // Pack constant array
     float _con[CEFFSSIZE] = {instrument->eta_inst * telescope->eta_fwd * telescope->eta_mir * 0.5,
         instrument->eta_inst * (1 - telescope->eta_fwd) * telescope->eta_mir * 0.5,
         instrument->eta_inst * (1 - telescope->eta_mir) * 0.5, instrument->eta_pb};

     float dt = 1. / instrument->freq_sample;
     
         // Copy constant array to Device constant memory
     gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL(const_effs), &_con, CEFFSSIZE * sizeof(float)) );
     gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL(cPI), &PI, sizeof(float)) );
     gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL(cCL), &CL, sizeof(float)) );
     gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL(cHP), &HP, sizeof(float)) );
     gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL(cKB), &KB, sizeof(float)) );
     gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL(cdt), &dt, sizeof(float)) );
     gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL(cfreq_chop), &(telescope->freq_chop), sizeof(float)) );
     gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL(cfreq_nod), &(telescope->freq_nod), sizeof(float)) );
     gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL(cfreq_sample), &(instrument->freq_sample), sizeof(float)) );
     gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL(cdAz_chop), &(telescope->dAz_chop), sizeof(float)) );
     gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL(cdelta), &(instrument->delta), sizeof(float)) );
     gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL(ct0), &(simparams->t0), sizeof(float)) );
     gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL(ch_column), &(atmosphere->h_column), sizeof(float)) );
     gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL(cv_wind), &(atmosphere->v_wind), sizeof(float)) );
    

     gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL(cnt), &(simparams->nTimes), sizeof(int)) );
     gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL(cnf_filt), &(instrument->nfreqs_filt), sizeof(int)) );
     gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL(cnf_src), &(source->nfreqs_src), sizeof(int)) );
     gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL(cnf_atm), &(atmosphere->nfreqs_atm), sizeof(int)) );
     gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL(cnAz), &(source->nAz), sizeof(int)) );
     gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL(cnEl), &(source->nEl), sizeof(int)) );
     gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL(cnPWV_atm), &(atmosphere->nPWV_atm), sizeof(int)) );
     gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL(cnx), &(atmosphere->nx), sizeof(int)) );
     gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL(cny), &(atmosphere->ny), sizeof(int)) );
     
     gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL(cOFF_empty), &(simparams->OFF_empty), sizeof(int)) );

     std::array<dim3, 2> BT;
     BT[0] = nrb;
     BT[1] = nrt;

     return BT;
}

/**
  Initialise an array of random states for pseudorandom number generation.
  Given a seed, each thread is assigned a sequence based on its index.
  Snippet taken from the CUDA handbook.

  @param state Array of hiprand states. Should be initialised and sized to total number of threads in grid.
  @param seed Integer describing the seed of the generator.
 */
__global__ void setup_kernel(hiprandState *state, unsigned long long int seed = 0) {
    if (!seed) {
        seed = clock64();
    }
    
    int idx = blockDim.x*blockIdx.x + threadIdx.x;
    hiprand_init(seed, idx, 0, &state[idx]);
}

/**
  Main simulation kernel. This is where the magic happens.

  @param I_atm Array containing blackbody intensity of atmosphere, in SI units.
  @param I_gnd Array containing blackbody intensity of ground, in SI units.
  @param I_tel Array containing blackbody intensity of telescope, in SI units.
  @param sigout Array for storing output power, for each channel, for each time, in SI units.
  @param flagout Array for storing wether beam is in chop A or B, in nod AB or BA.
  @param freqs_src Array containing bin frequencies, in Hertz.
  @param azsrc Array containing Azimuth co-ordinates of source, in degrees.
  @param elsrc Array containing Elevation co-ordinates of source, in degrees.
  @param eta_ap Array containing aperture efficiencies, for each bin frequency.
  @param x_atm Array containing x co-ordinates of the atmosphere screen, in meters.
  @param y_atm Array containing y co-ordinates of the atmosphere screen, in meters.
  @param PWV_screen Array containing PWV value of atmosphere, over the range described by x_atm and y_atm, in millimeters.
  @param freqs_atm Array containing frequencies over which to interpolate atmospheric transmission, in Hertz.
  @param PWV_atm Array containing PWV values over which to interpolate atmospheric transmission, in millimeters.
  @param eta_atm Array with transmission parameters as fuiunction of freqs_atm and PWV_atm.
  @param filterbank Array containing filterbank of instrument.
  @param source Array containing source intensity, as function of azsrc, elsrc and freqs_src, in SI units.
  @param state Array with states for drawing random Gaussian values for noise calculations.
 */
__global__ void runSimulation(float *I_atm, float *I_gnd, float *I_tel,
        float *sigout, float *azout, float *elout, int *flagout,
        float *freqs_src, float *azsrc, float *elsrc, float *eta_ap,
        float *x_atm, float *y_atm, float *PWV_screen, float *freqs_atm,
        float *PWV_atm, float *eta_atm, float *filterbank, float *source, hiprandState *state) {
    
    int idx = blockDim.x*blockIdx.x + threadIdx.x;

    if (idx < cnt)
    {
        float t_real; // Real time.
        float t_start; // Time from start of observation.
        float PWV_Gauss_interp; // Interpolated PWV of Gaussian smoothed screen.
        float eta_atm_interp; // Interpolated eta_atm, over frequency and PWV
        float freq; // Bin frequency
        float I_nu; // Specific intensity of source.
        float sigma_k; // Noise per channel.
        float eta_kj; // Filter efficiency for bin j, at channel k
        float sqrt_samp = sqrtf(0.5 * cfreq_sample); // Constant term needed for noise calculation
        int n_chop, n_nod, start_slice;
        int position;

        float dfreq = freqs_src[1] - freqs_src[0];
    
        hiprandState localState = state[idx];
        bool chop_flag;

        float is_in_lower_half;
        int nod_flag;

        AzEl center;

        center.Az = 0;
        center.El = 0;

        AzEl pointing;
        xy_atm point_atm;

        t_real = idx * cdt + ct0;
        t_start = idx * cdt;

        n_chop = floor(t_start * cfreq_chop);
        n_nod = floor(t_start * cfreq_nod);
        
        chop_flag = (n_chop % 2 != 0); // If even (false), ON. Odd (true), OFF.
        nod_flag = -1 + 2 * (n_nod % 2 != 0); // If even (false), AB. Odd (true), BA.
        
        is_in_lower_half = (t_start - n_nod / cfreq_nod) - (1 / cfreq_nod / 2);
        sgn(is_in_lower_half, position);
        position *= nod_flag;
        
        scanPoint(&center, &pointing, chop_flag, position * cdAz_chop);
        flagout[idx] = chop_flag * position + (1 - chop_flag) * (1 - position); 
        
        // STORAGE: Add current pointing to output array
        azout[idx] = pointing.Az;
        elout[idx] = pointing.El;

        convertAnglesToSpatialAtm(&pointing, &point_atm, ch_column);

        // Add wind to this - currently only along x-axis and pretty manual
        point_atm.xAz = point_atm.xAz + cv_wind * t_real;

        // Interpolate on PWV_Gauss
        PWV_Gauss_interp = interpValue(point_atm.xAz, point_atm.yEl, 
                x_atm, y_atm, cnx, cny, PWV_screen, 0);

        float* PSD_nu = new float[cnf_src];
        int chop_flag_sgn = abs(chop_flag);
        
        for(int j=0; j<cnf_src; j++)
        {   
            freq = freqs_src[j];
            eta_atm_interp = interpValue(freq, PWV_Gauss_interp, 
                    freqs_atm, PWV_atm, cnf_atm, cnPWV_atm, eta_atm, 0);

            start_slice = cnAz * cnEl * j;
            I_nu = interpValue(pointing.Az, pointing.El, 
                azsrc, elsrc, cnAz, cnEl, source, start_slice);
            
            //if (cOFF_empty && chop_flag_sgn) {

            //    I_nu = 0.;
            //}

            //else {
            //    I_nu = interpValue(pointing.Az, pointing.El, 
            //        azsrc, elsrc, cnAz, cnEl, source, start_slice);
            //}
            //if (idx == 0){
            //    printf("%d %d %.12e\n", cOFF_empty, chop_flag_sgn, I_nu);
            //}
            PSD_nu[j] = (eta_ap[j] * eta_atm_interp * const_effs[0] * I_nu
                + const_effs[0] * (1 - eta_atm_interp) * I_atm[j] 
                + const_effs[1] * I_gnd[j] 
                + const_effs[2] * I_tel[j]) * cCL*cCL / (freq*freq);
        }
        
        // In this loop, calculate P_k, NEP_k and noise
        for(int k=0; k<cnf_filt; k++) {
            float P_k = 0.; // Initialise each channel to zero, for each timestep
            float NEP_accum = 0.;

            // Can loop over bins again, cheap operations this time
            for(int j=0; j<cnf_src; j++) {   
                freq = freqs_src[j];
                eta_kj = filterbank[k*cnf_src + j];
                
                //printf("%.12e %d hallooooo\n", PSD_nu[j], j); 
                NEP_accum += PSD_nu[j] * eta_kj * (cHP * freq + PSD_nu[j] * eta_kj + 2 * cdelta / const_effs[3]);
                P_k += PSD_nu[j] * eta_kj;
            }

            sigma_k = sqrt(2 * NEP_accum * dfreq) * sqrt_samp;
            P_k *= dfreq;

            P_k += sigma_k * hiprand_normal(&localState);
            state[idx] = localState;
           
            // STORAGE: Add signal to signal array in output
            sigout[idx * cnf_filt + k] = P_k; 

        }
        delete[] PSD_nu;
    }
}

/**
  Run a TiEMPO2 simulation using CUDA.
 
  This function is exposed to the ctypes interface and can be called from Python..
 
  @param instrument CuInstrument object containing instrument to be simulated.
  @param telescope CuTelescope object containing telescope to be simulated.
  @param atmosphere CuAtmosphere object containing atmosphere parameters.
  @param source CuSource object containing source definitions.
  @param simparams CuSimParams object containing simulation parameters.
  @param output CuOutput object for storing simulation output.
 */
void runTiEMPO2_CUDA(CuInstrument *instrument, CuTelescope *telescope, CuAtmosphere *atmosphere, CuSource *source, 
        CuSimParams *simparams, CuOutput *output) {

    Timer timer;

    int nThreads = 256;
    int totalThreads = ceil(simparams->nTimes / nThreads) * nThreads;

    timer.start();

    
    std::array<dim3, 2> BT = initCUDA(instrument, telescope, simparams, source, atmosphere, nThreads);
    float freq;    // Frequency, used for initialising background sources.

    // Allocate and copy blackbodies
    float *I_atm = new float[source->nfreqs_src];
    float *I_gnd = new float[source->nfreqs_src];
    float *I_tel = new float[source->nfreqs_src];

    for(int j=0; j<source->nfreqs_src; j++)
    {
        freq = source->freqs_src[j];
        
        I_atm[j] = getPlanck(atmosphere->Tatm, freq); 
        I_gnd[j] = getPlanck(telescope->Tgnd, freq); 
        I_tel[j] = getPlanck(telescope->Ttel, freq);
    }
   
    float *dI_atm, *dI_gnd, *dI_tel;
    gpuErrchk( hipMalloc((void**)&dI_atm, source->nfreqs_src * sizeof(float)) );
    gpuErrchk( hipMalloc((void**)&dI_gnd, source->nfreqs_src * sizeof(float)) );
    gpuErrchk( hipMalloc((void**)&dI_tel, source->nfreqs_src * sizeof(float)) );

    gpuErrchk( hipMemcpy(dI_atm, I_atm, source->nfreqs_src * sizeof(float), hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy(dI_gnd, I_gnd, source->nfreqs_src * sizeof(float), hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy(dI_tel, I_tel, source->nfreqs_src * sizeof(float), hipMemcpyHostToDevice) );
    
    // Allocate and copy source arrays: freqs, az, el and I_nu
    float *dAz_src, *dEl_src, *dfreqs_src, *dI_nu;
    int nI_nu = source->nAz * source->nEl * source->nfreqs_src;

    gpuErrchk( hipMalloc((void**)&dAz_src, source->nAz * sizeof(float)) );
    gpuErrchk( hipMalloc((void**)&dEl_src, source->nEl * sizeof(float)) );
    gpuErrchk( hipMalloc((void**)&dfreqs_src, source->nfreqs_src * sizeof(float)) );
    gpuErrchk( hipMalloc((void**)&dI_nu, nI_nu * sizeof(float)) );
    
    gpuErrchk( hipMemcpy(dAz_src, source->Az, source->nAz * sizeof(float), hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy(dEl_src, source->El, source->nEl * sizeof(float), hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy(dfreqs_src, source->freqs_src, source->nfreqs_src * sizeof(float), hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy(dI_nu, source->I_nu, nI_nu * sizeof(float), hipMemcpyHostToDevice) );
    
    // Allocate and copy telescope arrays
    float *deta_ap;
    gpuErrchk( hipMalloc((void**)&deta_ap, source->nfreqs_src * sizeof(float)) );
    gpuErrchk( hipMemcpy(deta_ap, telescope->eta_ap, source->nfreqs_src * sizeof(float), hipMemcpyHostToDevice) );

    // Allocate and copy atmosphere arrays
    float *dx_atm, *dy_atm, *dPWV_screen, *dfreqs_atm, *dPWV_atm, *deta_atm;
    int nPWV_screen = atmosphere->nx * atmosphere->ny;
    int neta_atm = atmosphere->nfreqs_atm * atmosphere->nPWV_atm;
    
    gpuErrchk( hipMalloc((void**)&dx_atm, atmosphere->nx * sizeof(float)) );
    gpuErrchk( hipMalloc((void**)&dy_atm, atmosphere->ny * sizeof(float)) );
    gpuErrchk( hipMalloc((void**)&dPWV_screen, nPWV_screen * sizeof(float)) );
    gpuErrchk( hipMalloc((void**)&dfreqs_atm, atmosphere->nfreqs_atm * sizeof(float)) );
    gpuErrchk( hipMalloc((void**)&dPWV_atm, atmosphere->nPWV_atm * sizeof(float)) );
    gpuErrchk( hipMalloc((void**)&deta_atm, neta_atm * sizeof(float)) );

    gpuErrchk( hipMemcpy(dx_atm, atmosphere->x_atm, atmosphere->nx * sizeof(float), hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy(dy_atm, atmosphere->y_atm, atmosphere->ny * sizeof(float), hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy(dPWV_screen, atmosphere->PWV, nPWV_screen * sizeof(float), hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy(dfreqs_atm, atmosphere->freqs_atm, atmosphere->nfreqs_atm * sizeof(float), hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy(dPWV_atm, atmosphere->PWV_atm, atmosphere->nPWV_atm * sizeof(float), hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy(deta_atm, atmosphere->eta_atm, neta_atm * sizeof(float), hipMemcpyHostToDevice) );

    // Allocate and copy instrument arrays
    float *dfilterbank;
    int nfilterbank = source->nfreqs_src * instrument->nfreqs_filt;
    gpuErrchk( hipMalloc((void**)&dfilterbank, nfilterbank * sizeof(float)) );
    
    gpuErrchk( hipMemcpy(dfilterbank, instrument->filterbank, nfilterbank * sizeof(float), hipMemcpyHostToDevice) );

    // Allocate output arrays
    float *sigout;
    gpuErrchk( hipMalloc((void**)&sigout, (source->nfreqs_src * simparams->nTimes) * sizeof(float)) );
    float *azout, *elout;
    gpuErrchk( hipMalloc((void**)&azout, simparams->nTimes * sizeof(float)) );
    gpuErrchk( hipMalloc((void**)&elout, simparams->nTimes * sizeof(float)) );
    
    int *flagout;
    gpuErrchk( hipMalloc((void**)&flagout, simparams->nTimes * sizeof(int)) );

    // Setup rng for noise calcs
    hiprandState *devStates;
    gpuErrchk( hipMalloc((void **)&devStates, totalThreads * sizeof(hiprandState)) );
    setup_kernel<<<BT[0], BT[1]>>>(devStates);
    
    gpuErrchk( hipDeviceSynchronize() );
    
    // Set total heap reservation to 128 mb
    gpuErrchk( hipDeviceSetLimit(hipLimitMallocHeapSize, 128*1024*1024) );
    
    timer.stop();
    output->t_diag[0] = timer.get();
    
    // CALL TO MAIN SIMULATION KERNEL
    timer.start();

    runSimulation<<<BT[0], BT[1]>>>(dI_atm, dI_gnd, dI_tel,
            sigout, azout, elout, flagout,
            dfreqs_src, dAz_src, dEl_src, deta_ap, dx_atm, dy_atm, dPWV_screen, dfreqs_atm,
            dPWV_atm, deta_atm, dfilterbank, dI_nu, devStates);

    gpuErrchk( hipDeviceSynchronize() );
    
    timer.stop();
    output->t_diag[1] = timer.get();
    
    timer.start();

    gpuErrchk( hipMemcpy(output->signal, sigout, (instrument->nfreqs_filt * simparams->nTimes) * sizeof(float), hipMemcpyDeviceToHost) );

    gpuErrchk( hipMemcpy(output->Az, azout, simparams->nTimes * sizeof(int), hipMemcpyDeviceToHost) );
    gpuErrchk( hipMemcpy(output->El, elout, simparams->nTimes * sizeof(int), hipMemcpyDeviceToHost) );
    
    gpuErrchk( hipMemcpy(output->flag, flagout, simparams->nTimes * sizeof(int), hipMemcpyDeviceToHost) );

    gpuErrchk( hipDeviceReset() );

    delete[] I_atm;
    delete[] I_gnd;
    delete[] I_tel;
    
    timer.stop();
    output->t_diag[2] = timer.get();
}

