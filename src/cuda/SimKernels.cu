#include "hip/hip_runtime.h"
#include "InterfaceCUDA.h"

/*! \file Kernels.cu
    \brief Definitions of CUDA kernels for TiEMPO2.

    author: Arend Moerman
*/

// OBSERVATION-INSTRUMENT PARAMETERS
__constant__ float const_effs[CEFFSSIZE];   // Contains constant efficiencies:chain, gnd, mir, pb 
__constant__ float cdt;                     // Timestep
__constant__ float cfreq_chop;              // Chopping frequency
__constant__ float cfreq_nod;               // Nodding frequency
__constant__ float cf_sample;            // Sampling frequency of readout
__constant__ float cdAz_chop;               // Chopping throw
__constant__ float cdelta;                  // Bandgap energy of MKID
__constant__ int cnt;                       // Number of time evals
__constant__ int cnf_ch;                    // Number of filter freqs
__constant__ int cchop_mode;                // What chopping scheme to use

// ATMOSPHERE PARAMETERS
__constant__ float ch_column;               // Column height
__constant__ float cv_wind;                 // Windspeed

// SCAN PARAMETERS
__constant__ int cscantype;
__constant__ float ccscEl0;
__constant__ float cAx;
__constant__ float cAxmin;
__constant__ float cAy;
__constant__ float cAymin;
__constant__ float cwx;
__constant__ float cwxmin;
__constant__ float cwy;
__constant__ float cwymin;
__constant__ float cphix;
__constant__ float cphiy;

// TEXTURE MEMORY
texture<float, hipTextureType1D, hipReadModeElementType> tex_filterbank;
texture<float, hipTextureType1D, hipReadModeElementType> tex_eta_ap_ON;
texture<float, hipTextureType1D, hipReadModeElementType> tex_eta_ap_OFF;
texture<float, hipTextureType1D, hipReadModeElementType> tex_I_atm;
texture<float, hipTextureType1D, hipReadModeElementType> tex_I_gnd;
texture<float, hipTextureType1D, hipReadModeElementType> tex_I_tel;
texture<float, hipTextureType1D, hipReadModeElementType> tex_I_CMB;

#define KB              1.380649E-23f
#define CL              2.9979246E8f
#define HP              6.62607015E-34f

#define NTHREADS1D      256
#define NTHREADS2DX     32
#define NTHREADS2DY     16

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }

/**
  Check CUDA API error status of call.
 
  Wrapper for finding errors in CUDA API calls.
 
  @param code The errorcode returned from failed API call.
  @param file The file in which failure occured.
  @param line The line in file in which error occured.
  @param abort Exit code upon error.
 */
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

/**
  Write a CUDA device array to a file, for debugging.

  @param array Pointer to device array of type T.
  @param s_array Size of array.
  @param name_txt Name of file to write array to. Name is appended with '.txt' by the function itself.
 */
template <typename T>
__host__ void writeArray(T *array, int s_array, std::string name_txt) {
    
    T *h_array = new T[s_array];
    gpuErrchk( hipMemcpy(h_array, array, s_array * sizeof(T), hipMemcpyDeviceToHost) );
    
    std::ofstream myfile (name_txt + ".txt");
    if (myfile.is_open())
    {
        for(int count = 0; count < s_array; count ++){
            myfile << h_array[count] << "\n" ;
        }

        myfile.close();
    }
    else std::cout << "Unable to open file";
    delete[] h_array;
}

/**
  Calculate Planckian distribution.

  Used for calculating blackbody intensities of atmosphere, ground and telescope.

  @param T Temperature of blackbody, in Kelvin.
  @param nu Frequency at which to evaluate blackbody, in Hertz.

  @returns Blackbody intensity.
 */
__host__ float getPlanck(float T, float nu)
{
    float prefac = 2 * HP * nu*nu*nu / (CL*CL);
    float dist = 1 / (exp(HP*nu / (KB*T)) - 1);
    
    return prefac * dist;
}

/**
  Calculate sign of number.

  Used in determining chop-nod state.

  @param val Value of number.
 */
__device__ __inline__ void sgn(float val, int &out) {
    out = (float(0) < val) - (val < float(0));
}

/**
  Convert angle in arcseconds to degrees.

  @param ang angle in arcseconds.
 */
__host__ __device__ __inline__ float as2deg(float ang) {
    return ang / 3600;
}

/**
  Convert angle in degrees to radian.

  @param ang Angle in degrees.
 */
__host__ __device__ __inline__ float deg2rad(float ang) {
    return ang * 0.017453292;
}

/**
  Calculate new Azimuth-Elevation co-ordinate, accoding to chop position.

  This function just "scans" a single point, so seems sort of pointless. 
  Still implemented for completeness.

  @param center Az-El co-ordinate of point to observe, w.r.t. source Az-El.
  @param out Container for storing output Az-El co-ordinate.
  @param chop Whether chopper is in A (false) or B (true).
  @param sep Angular throw between chop A and B, in degrees.
 */
__device__ __inline__ void scanPoint(AzEl* center, AzEl* out, bool chop, float sep = 0.) {
    float offset = 0.;
    
    if (chop) {
        offset = sep;
    }    
    out->Az = center->Az + offset;
    out->El = center->El;
}

__device__ __inline__ void scanDaisy(AzEl* center, AzEl* out, float t, bool chop, float sep = 0.) {
    float offset = 0.;
    
    if (chop) {
        offset = sep;
    }    
    
    out->Az = center->Az + offset + cAx*sinf(cwx*t)*cosf(cwx*t + deg2rad(cphix)) + cAxmin*sinf(cwxmin*t)*cosf(cwxmin*t + deg2rad(cphix));
    out->El = center->El + cAy*sinf(cwy*t)*sinf(cwy*t + deg2rad(cphiy)) + cAymin*sinf(cwymin*t)*sinf(cwymin*t + deg2rad(cphiy)) - cAy;
}

/**
  Convert an Az-El co-ordinate to a projected x-y co-ordinate on the atmosphere.

  @param angles Az-El co-ordinate to convert.
  @param out Container for storing the calculated x-y point.
 */
__device__ __inline__ void convertAnglesToSpatialAtm(AzEl* angles, xy_atm* out) {
    
    float coord = tanf(deg2rad(angles->Az)) * ch_column;
    
    out->xAz = coord;
    coord = tanf(deg2rad(angles->El)) * ch_column;
    out->yEl = coord;
}

__device__ __inline__ void getABBA_posflag(float &t_start, AzEl *center, AzEl *pointing, int &flagout) {
    int n_chop;
    int n_nod;
    int position;

    bool chop_flag;

    float is_in_lower_half;
    int nod_flag;

    n_chop = floorf(t_start * cfreq_chop);
    n_nod = floorf(t_start * cfreq_nod);
    
    chop_flag = (n_chop % 2 != 0); // If even (false), ON. Odd (true), OFF.
    nod_flag = -1 + 2 * (n_nod % 2 != 0); // If even (false), AB. Odd (true), BA.
    
    is_in_lower_half = (t_start - n_nod / cfreq_nod) - (1 / cfreq_nod / 2);
    sgn(is_in_lower_half, position);
    position *= nod_flag;
    
    scanPoint(center, pointing, chop_flag, position * cdAz_chop);
    flagout = chop_flag * position + (1 - chop_flag) * (1 - position);
}

__device__ __inline__ void getONOFF_posflag(float &t_start, AzEl *center, AzEl *pointing, int &flagout) {
    int n_chop;
    bool chop_flag;

    n_chop = floorf(t_start * cfreq_chop);
    
    chop_flag = (n_chop % 2 != 0); // If even (false), ON. Odd (true), OFF.
    if(cscantype == 0) {scanPoint(center, pointing, chop_flag, cdAz_chop);}
    else if(cscantype == 1) {scanDaisy(center, pointing, t_start, chop_flag, cdAz_chop);}
    flagout = chop_flag;
}

__device__ __inline__ void getnochop_posflag(float &t_start, AzEl *center, AzEl *pointing, int &flagout) {
    if(cscantype == 0) {scanPoint(center, pointing, 0, cdAz_chop);}
    else if(cscantype == 1) {scanDaisy(center, pointing, t_start, 0, cdAz_chop);}
    flagout = 0;
}

/**
  Initialize CUDA.
 
  Instantiate program and populate constant memory.
 
  @param instrument CuInstrument object containing instrument to be simulated.
  @param telescope CuTelescope object containing telescope to be simulated.
  @param source CuSource object containing source definitions.
  @param atmosphere CuAtmosphere object containing atmosphere parameters.
  @param nTimes number of time evaluations in simulation.

  @return BT Array of two dim3 objects, containing number of blocks per grid and number of threads per block.
 */
__host__ void initCUDA(Instrument<float> *instrument, Telescope<float> *telescope, Source<float> *source, Atmosphere<float> *atmosphere, int nTimes) {
    // Pack constant array
    float _con[CEFFSSIZE] = {instrument->eta_inst * instrument->eta_misc * telescope->eta_fwd * telescope->eta_mir * 0.5,
        instrument->eta_inst * instrument->eta_misc * (1 - telescope->eta_fwd) * telescope->eta_mir * 0.5,
        instrument->eta_inst * instrument->eta_misc * (1 - telescope->eta_mir) * 0.5, 
        instrument->eta_pb};

    float dt = 1. / instrument->f_sample;
     
    // OBSERVATION-INSTRUMENT PARAMETERS
    gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL(const_effs), &_con, CEFFSSIZE * sizeof(float)) );
    gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL(cdt), &dt, sizeof(float)) );
    gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL(cfreq_chop), &(telescope->freq_chop), sizeof(float)) );
    gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL(cfreq_nod), &(telescope->freq_nod), sizeof(float)) );
    gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL(cf_sample), &(instrument->f_sample), sizeof(float)) );
    gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL(cdAz_chop), &(telescope->dAz_chop), sizeof(float)) );
    gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL(cdelta), &(instrument->delta), sizeof(float)) );
    gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL(cnt), &nTimes, sizeof(int)) );
    gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL(cnf_ch), &(instrument->nf_ch), sizeof(int)) );
    gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL(cchop_mode), &(telescope->chop_mode), sizeof(int)) );
    
    // ATMOSPHERE PARAMETERS
    gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL(ch_column), &(atmosphere->h_column), sizeof(float)) );
    gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL(cv_wind), &(atmosphere->v_wind), sizeof(float)) );

    // SCAN PARAMETERS
    float cscEl0 = 1. / sinf(deg2rad(telescope->El0));

    gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL(cscantype), &(telescope->scantype), sizeof(int)) );
    gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL(ccscEl0), &cscEl0, sizeof(float)) );
    gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL(cAx), &(telescope->Ax), sizeof(float)) );
    gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL(cAxmin), &(telescope->Axmin), sizeof(float)) );
    gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL(cAy), &(telescope->Ay), sizeof(float)) );
    gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL(cAymin), &(telescope->Aymin), sizeof(float)) );
    gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL(cwx), &(telescope->wx), sizeof(float)) );
    gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL(cwxmin), &(telescope->wxmin), sizeof(float)) );
    gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL(cwy), &(telescope->wy), sizeof(float)) );
    gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL(cwymin), &(telescope->wymin), sizeof(float)) );
    gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL(cphix), &(telescope->phix), sizeof(float)) );
    gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL(cphiy), &(telescope->phiy), sizeof(float)) );
}

/**
  Obtain timestream for chopping states, PWV from atmosphere, and rng for Gaussian draw.
  Each timestep gets a new seed for the Gaussian, just to randomise it even harder.
  
  @param state Array of hiprand states. Should be initialised and sized to total number of threads in grid.
  @param seed Integer describing the seed of the generator.
 */
__global__ void get_chop_pwv_rng(ArrSpec<float> Az_spec, ArrSpec<float> El_spec, 
                                 ArrSpec<float> x_atm, ArrSpec<float> y_atm, 
                                 float *center, float *PWV_screen, float *PWV_out, 
                                 int *flagout, float *azout, float *elout,
                                 hiprandState *state, unsigned long long int seed = 0) {
    if (!seed) {
        seed = clock64();
    }

    int idx = blockIdx.x * blockDim.x + threadIdx.x; 
    
    if (idx < cnt) {

        // FLOATS
        float time;         // Timepoint for thread in simulation.
        float _PWV_out;     // Container for storing interpolated PWV values.

        // INTEGERS
        int flag;           // Flag for storing chop position.

        // CUSTOM STRUCTS
        AzEl pointing;      // Struct for storing the current pointing (w.r.t. center_p).
        AzEl center_p;      // Struct for storing the central pointing.
        xy_atm point_atm;   // Struct for storing projected pointing x-y coordinates on atmosphere screen.
        
        center_p.Az = center[0];
        center_p.El = center[1];

        time = idx * cdt;
        
        if(cchop_mode == 0) {getnochop_posflag(time, &center_p, &pointing, flag);}
        else if(cchop_mode == 1) {getONOFF_posflag(time, &center_p, &pointing, flag);}
        else if(cchop_mode == 2) {getABBA_posflag(time, &center_p, &pointing, flag);}

        convertAnglesToSpatialAtm(&pointing, &point_atm);

        // Add wind to this - currently only along x-axis and pretty manual
        point_atm.xAz = point_atm.xAz + cv_wind * time;
    
        interpValue(point_atm.xAz, point_atm.yEl,
                    &x_atm, &y_atm, PWV_screen, 0, _PWV_out);            
    
        __syncthreads();
        hiprand_init(seed, idx, 0, &state[idx]);
        azout[idx] = pointing.Az;
        elout[idx] = pointing.El;
        flagout[idx] = flag;
        
        PWV_out[idx] = _PWV_out;
    }
}

__device__ void commonJob(ArrSpec<float> *f_src, ArrSpec<float> *f_atm, ArrSpec<float> *PWV_atm, int idx, int idy, float *sigout, float *nepout,
        float *PWV_trace, float *eta_atm, float I_nu, int flag) {
        
    // FLOATS
    float eta_atm_interp;   // Interpolated eta_atm, over frequency and PWV
    float freq;             // Bin frequency
    float PSD_nu;           // Local variable for storing PSD.
    float eta_kj;           // Filter efficiency for bin j, at channel k.
    float PWV_tr;           // Local variable for storing PWV value at time.
    float eta_ap;           // Local variable for storing aperture efficiency
    float sigfactor;        // Factor for calculating power. Perform outside of channel loop for speed.
    float nepfactor1;       // Factor 1 for calculating NEP. Perform outside of channel loop for speed.
    float nepfactor2;       // Factor 2 for calculating NEP. Perform outside of channel loop for speed.

    // Reusable symbols for interpolation stuff - listed separately for readability
    PWV_tr = PWV_trace[idx];

    freq = f_src->start + f_src->step * idy;

    interpValue(PWV_tr, freq,
                PWV_atm, f_atm,
                eta_atm, 0, eta_atm_interp);

    if(flag == 0 or flag == -1) {
        eta_ap = tex1Dfetch(tex_eta_ap_ON, idy); 
    }

    else {
        eta_ap = tex1Dfetch(tex_eta_ap_OFF, idy);
    }

    eta_atm_interp = powf(eta_atm_interp, ccscEl0);

    PSD_nu = eta_ap * eta_atm_interp * const_effs[0] * I_nu
        + ( const_effs[0] * (1 - eta_atm_interp) * tex1Dfetch(tex_I_atm, idy)
        + const_effs[1] * tex1Dfetch(tex_I_gnd, idy)
        + const_effs[2] * tex1Dfetch(tex_I_tel, idy)) 
        * CL*CL / (freq*freq);

    sigfactor = PSD_nu * f_src->step;
    nepfactor1 = sigfactor * (HP * freq + 2 * cdelta / const_effs[3]);
    nepfactor2 = sigfactor * PSD_nu;

    #pragma unroll 
    for(int k=0; k<cnf_ch; k++) {
        eta_kj = tex1Dfetch( tex_filterbank, k*f_src->num + idy);
        atomicAdd(&sigout[k*cnt + idx], __fmul_rn(eta_kj, sigfactor)); 
        atomicAdd(&nepout[k*cnt + idx], __fmul_rn(eta_kj, __fmaf_rn(nepfactor2, eta_kj, nepfactor1))); 
    }
}

/**
  Calculate power and NEP in each channel.
  This kernel is optimised for strict single-pointing (i.e., no sky-chopping) observations.

  @param sigout Array for storing output power, for each channel, for each time, in SI units.
  @param nepout Array for storing output NEP, for each channel, for each time, in SI units.
  @param flagout Array for storing wether beam is in chop A or B, in nod AB or BA.
  @param PWV_trace Array containing PWV value of atmosphere as seen by telescope over observation, in millimeters.
  @param eta_atm Array with transmission parameters as function of PWV and frequency.
  @param source Array containing source intensity at three pointings, as function of frequency, in SI units.
 */
__global__ void calcPowerNEP_1(ArrSpec<float> f_src, ArrSpec<float> f_atm, ArrSpec<float> PWV_atm, float *sigout, float *nepout, int *flagout,
        float *PWV_trace, float *eta_atm, float *source) {
    
    int idx = blockIdx.x * blockDim.x + threadIdx.x; 
    int idy = blockIdx.y * blockDim.y + threadIdx.y; 

    if (idx < cnt && idy < f_src.num) {
        float I_nu;             // Specific intensity of source.

        // INTEGERS
        int flag = flagout[idx];

        I_nu = source[idy];

        commonJob(&f_src, &f_atm, &PWV_atm, idx, idy, sigout, nepout, PWV_trace, eta_atm, I_nu, flag);
    }
}

/**
  Calculate power and NEP in each channel.
  This kernel is optimised for observations involving two pointings, encountered in single-pointing ON-OFF chopping.
        
  @param sigout Array for storing output power, for each channel, for each time, in SI units.
  @param nepout Array for storing output NEP, for each channel, for each time, in SI units.
  @param flagout Array for storing wether beam is in chop A or B, in nod AB or BA.
  @param PWV_trace Array containing PWV value of atmosphere as seen by telescope over observation, in millimeters.
  @param eta_atm Array with transmission parameters as function of PWV and frequency.
  @param source Array containing source intensity at three pointings, as function of frequency, in SI units.
 */
__global__ void calcPowerNEP_2(ArrSpec<float> f_src, ArrSpec<float> f_atm, ArrSpec<float> PWV_atm, float *sigout, float *nepout, int *flagout,
        float *PWV_trace, float *eta_atm, float *source) {
    
    int idx = blockIdx.x * blockDim.x + threadIdx.x; 
    int idy = blockIdx.y * blockDim.y + threadIdx.y; 

    if (idx < cnt && idy < f_src.num) {
        float I_nu;             // Specific intensity of source.
        int idx_point;          // Index for pointing of source (0 is OFF-B, 1 is ON-AB, 2 is OFF-A)

        // INTEGERS
        int flag = flagout[idx];
        
        // Determine idx in source from chopping flag
        if(flag==0) {idx_point = 0;}
        else {idx_point = 1;}
    
        I_nu = source[idx_point*f_src.num + idy];

        commonJob(&f_src, &f_atm, &PWV_atm, idx, idy, sigout, nepout, PWV_trace, eta_atm, I_nu, flag);
    }
}

/**
  Calculate power and NEP in each channel.
  This kernel is optimised for observations involving three pointings, encountered in single-pointing ABBA chopping.
        
  @param sigout Array for storing output power, for each channel, for each time, in SI units.
  @param nepout Array for storing output NEP, for each channel, for each time, in SI units.
  @param flagout Array for storing wether beam is in chop A or B, in nod AB or BA.
  @param PWV_trace Array containing PWV value of atmosphere as seen by telescope over observation, in millimeters.
  @param eta_atm Array with transmission parameters as function of PWV and frequency.
  @param source Array containing source intensity at three pointings, as function of frequency, in SI units.
 */
__global__ void calcPowerNEP_3(ArrSpec<float> f_src, ArrSpec<float> f_atm, ArrSpec<float> PWV_atm, float *sigout, float *nepout, int *flagout,
        float *PWV_trace, float *eta_atm, float *source) {
    
    int idx = blockIdx.x * blockDim.x + threadIdx.x; 
    int idy = blockIdx.y * blockDim.y + threadIdx.y; 

    if (idx < cnt && idy < f_src.num) {
        float I_nu;             // Specific intensity of source.
        int idx_point;          // Index for pointing of source (0 is OFF-B, 1 is ON-AB, 2 is OFF-A)

        //printf("%d\n", idx);
        // INTEGERS
        int flag = flagout[idx];
        
        // Determine idx in source from chopping flag
        if(flag==0 or flag==2) {idx_point = 1;}
        else if(flag==1) {idx_point = 2;}
        else {idx_point = 0;}
    
        I_nu = source[idx_point*f_src.num + idy];

        commonJob(&f_src, &f_atm, &PWV_atm, idx, idy, sigout, nepout, PWV_trace, eta_atm, I_nu, flag);
    }
}

/**
  Main simulation kernel. This is where the magic happens.

  @param sigout Array for storing output power, for each channel, for each time, in SI units.
  @param nepout Array for storing output NEP, for each channel, for each time, in SI units.
  @param azout Array containing Azimuth coordinates as function of time.
  @param elout Array containing Elevation coordinates as function of time.
  @param flagout Array for storing wether beam is in chop A or B, in nod AB or BA.
  @param PWV_trace Array containing PWV value of atmosphere as seen by telescope over observation, in millimeters.
  @param eta_atm Array with transmission parameters as function of freqs_atm and PWV_atm.
  @param source Array containing source intensity, as function of azsrc, elsrc and freqs_src, in SI units.
 */
__global__ void calcPowerNEP(ArrSpec<float> f_src, ArrSpec<float> f_atm, ArrSpec<float> PWV_atm, ArrSpec<float> Az_src, ArrSpec<float> El_src, float *sigout, float *nepout, float *azout, float *elout, int *flagout,
        float *PWV_trace, float *eta_atm, float *source) {
    
    int idx = blockIdx.x * blockDim.x + threadIdx.x; 
    int idy = blockIdx.y * blockDim.y + threadIdx.y; 

    if (idx < cnt && idy < f_src.num) {
        float I_nu;             // Specific intensity of source.
        // Reusable symbols for interpolation stuff
        int x0y0, x1y0, x0y1, x1y1;
        float t, u;
            
        AzEl pointing;

        pointing.Az = azout[idx];
        pointing.El = elout[idx];

        int iAz = floorf((pointing.Az - Az_src.start) / Az_src.step);
        int iEl = floorf((pointing.El - El_src.start) / El_src.step);

        int flag = flagout[idx];

        float Az_src_max = Az_src.start + Az_src.step * (Az_src.num - 1);
        float El_src_max = El_src.start + El_src.step * (El_src.num - 1);

        bool offsource = ((pointing.Az < Az_src.start) or (pointing.Az > Az_src_max)) or 
                         ((pointing.El < El_src.start) or (pointing.El > El_src_max));

        if(offsource) {I_nu = tex1Dfetch(tex_I_CMB, idy);}
        
        else {
            x0y0 = f_src.num * (iAz + iEl * Az_src.num);
            x1y0 = f_src.num * (iAz + 1 + iEl * Az_src.num);
            x0y1 = f_src.num * (iAz + (iEl+1) * Az_src.num);
            x1y1 = f_src.num * (iAz + 1 + (iEl+1) * Az_src.num);
            
            t = (pointing.Az - (Az_src.start + Az_src.step*iAz)) / Az_src.step;
            u = (pointing.El - (El_src.start + El_src.step*iEl)) / El_src.step;
            
            I_nu = (1-t)*(1-u) * source[x0y0 + idy];
            I_nu += t*(1-u) * source[x1y0 + idy];
            I_nu += (1-t)*u * source[x0y1 + idy];
            I_nu += t*u * source[x1y1 + idy];
        }

        commonJob(&f_src, &f_atm, &PWV_atm, idx, idy, sigout, nepout, PWV_trace, eta_atm, I_nu, flag);
    }
}

/**
  Calculate the total photon noise in a filter channel.

  After calculating the noise std from the NEP, a random number from a Gaussian is drawn and added to the total power in a channel.
  Note that, because we do not need the NEP after this step, we replace the value with a random Gaussian. 
  This is necessary for the TLS noise calculation, which comes after.

  @param sigout Array for storing output power, for each channel, for each time, in SI units.
  @param nepout Array for storing output NEP, for each channel, for each time, in SI units.
  @param state Array with states for drawing random Gaussian values for noise calculations.
 */
__global__ void calcPhotonNoise(float *sigout, float *nepout, hiprandState *state) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x; 
    if (idx < cnt) {
        hiprandState localState = state[idx];
        float sqrt_samp = sqrtf(0.5 * cf_sample); // Constant term needed for noise calculation
        float sigma_k, P_k;


        for(int k=0; k<cnf_ch; k++) {
            sigma_k = sqrtf(2 * nepout[k*cnt + idx]) * sqrt_samp;
            P_k = sigma_k * hiprand_normal(&localState);

            state[idx] = localState;

            sigout[k*cnt + idx] += P_k;

            nepout[k*cnt + idx] = hiprand_normal(&localState);
            state[idx] = localState;
        }
    }
}

/**
  Run a TiEMPO2 simulation using CUDA.
 
  This function is exposed to the ctypes interface and can be called from Python..
 
  @param instrument CuInstrument object containing instrument to be simulated.
  @param telescope CuTelescope object containing telescope to be simulated.
  @param atmosphere CuAtmosphere object containing atmosphere parameters.
  @param source CuSource object containing source definitions.
  @param output CuOutput object for storing simulation output.
  @param nTimes Number of time evaluations in simulation.
 */
void runTiEMPO2_CUDA(Instrument<float> *instrument, Telescope<float> *telescope, Atmosphere<float> *atmosphere, Source<float> *source, Output<float> *output, int nTimesTotal, char *outpath) {
    // FLOATS
    float *d_sigout;        // Device pointer for output power array
    float *d_nepout;        // Device pointer for output NEP array
    float *d_azout;         // Device pointer for output Azimuth array 
    float *d_elout;         // Device pointer for output Elevation array
    float *d_I_nu;          // Device pointer for source intensities
    
    // INTEGERS
    int *d_flagout;         // Device pointer for output chopping flags
    int nffnt;              // Number of filter frequencies times number of time evaluations
    int nf_src;             // Number of frequency points in source.
    int numSMs;             // Number of streaming multiprocessors on GPU
    int nBlocks1D;          // Number of 1D blocks, in terms of number of SMs
    int nBlocks2Dx;         // Number of 2D blocks, in terms of number of SMs, along the x-dimension
    int nBlocks2Dy;         // Number of 2D blocks, in terms of number of SMs, along the y-dimension

    // OTHER DECLARATIONS
    dim3 blockSize1D;       // Size of 1D block (same as nThreads1D, but dim3 type)
    dim3 gridSize1D;        // Number of 1D blocks per grid
    dim3 blockSize2D;       // Size of 2D block, along x and y dimension
    dim3 gridSize2D;        // Number of 2D blocks per grid
    Timer timer;            // Timer class for timing kernel invocations

    // ALLOCATE ARRAY SPECIFICATION COPIES
    struct ArrSpec<float> _f_spec = instrument->f_spec;
    struct ArrSpec<float> _Az_src = source->Az_spec;
    struct ArrSpec<float> _El_src = source->El_spec;
    
    struct ArrSpec<float> _f_atm;
    struct ArrSpec<float> _PWV_atm;
    float *eta_atm;


    readEtaATM<float, ArrSpec<float>>(&eta_atm, &_PWV_atm, &_f_atm);
    
    std::string str_path(atmosphere->path);
    std::string str_outpath(outpath);

    int *meta;
    readAtmMeta(&meta, str_path);

    // Calculate lengths of x and y of single screen
    float lx = meta[1]*atmosphere->dx;
    float ly = meta[2]*atmosphere->dy;
    float lx_av = lx - ly;
    float t_obs_av = lx_av / atmosphere->v_wind; // Max available time per screen

    float timeTotal = nTimesTotal / instrument->f_sample;

    int nJobs = ceil(timeTotal / t_obs_av);
    int nTimesScreen = floor(t_obs_av * instrument->f_sample); // If error, change ceil to floor

    struct ArrSpec<float> _x_atm;
    struct ArrSpec<float> _y_atm;

    _x_atm.start = -ly/2;
    _x_atm.step = atmosphere->dx;
    _x_atm.num = meta[1];
    
    _y_atm.start = -ly/2;
    _y_atm.step = atmosphere->dy;
    _y_atm.num = meta[2];

    // Initialize constant memory
    initCUDA(instrument, telescope, source, atmosphere, nTimesScreen); 

    
    nf_src = _f_spec.num; // Number of spectral points in source
    
    gpuErrchk( hipDeviceGetAttribute(&numSMs, hipDeviceAttributeMultiprocessorCount, 0) );

    // TiEMPO2 prefers larger L1 cache over shared memory.
    gpuErrchk( hipDeviceSetCacheConfig(hipFuncCachePreferL1) );

    timer.start();
    

    float freq;    // Frequency, used for initialising background sources.

    // Allocate and copy blackbodies
    std::vector<float> I_atm(nf_src);
    std::vector<float> I_gnd(nf_src);
    std::vector<float> I_tel(nf_src);
    std::vector<float> I_CMB(nf_src);

    for(int j=0; j<nf_src; j++)
    {
        freq = _f_spec.start + _f_spec.step * j;
        
        I_atm[j] = getPlanck(atmosphere->Tatm, freq); 
        I_gnd[j] = getPlanck(telescope->Tgnd, freq); 
        I_tel[j] = getPlanck(telescope->Ttel, freq);
        I_CMB[j] = getPlanck(2.725, freq);
    }
    
    float *dI_atm, *dI_gnd, *dI_tel, *dI_CMB;
    
    gpuErrchk( hipMalloc((void**)&dI_atm, nf_src * sizeof(float)) );
    gpuErrchk( hipMalloc((void**)&dI_gnd, nf_src * sizeof(float)) );
    gpuErrchk( hipMalloc((void**)&dI_tel, nf_src * sizeof(float)) );
    gpuErrchk( hipMalloc((void**)&dI_CMB, nf_src * sizeof(float)) );

    gpuErrchk( hipMemcpy(dI_atm, I_atm.data(), nf_src * sizeof(float), hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy(dI_gnd, I_gnd.data(), nf_src * sizeof(float), hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy(dI_tel, I_tel.data(), nf_src * sizeof(float), hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy(dI_CMB, I_CMB.data(), nf_src * sizeof(float), hipMemcpyHostToDevice) );
    
    gpuErrchk( hipBindTexture((size_t)0, tex_I_atm, dI_atm, nf_src * sizeof(float)) );
    gpuErrchk( hipBindTexture((size_t)0, tex_I_gnd, dI_gnd, nf_src * sizeof(float)) );
    gpuErrchk( hipBindTexture((size_t)0, tex_I_tel, dI_tel, nf_src * sizeof(float)) );
    gpuErrchk( hipBindTexture((size_t)0, tex_I_CMB, dI_CMB, nf_src * sizeof(float)) );
    
    
    // Allocate and copy telescope arrays
    float *deta_ap_ON, *deta_ap_OFF;
    gpuErrchk( hipMalloc((void**)&deta_ap_ON, nf_src * sizeof(float)) );
    gpuErrchk( hipMalloc((void**)&deta_ap_OFF, nf_src * sizeof(float)) );
    gpuErrchk( hipMemcpy(deta_ap_ON, telescope->eta_ap_ON, nf_src * sizeof(float), hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy(deta_ap_OFF, telescope->eta_ap_OFF, nf_src * sizeof(float), hipMemcpyHostToDevice) );
    
    gpuErrchk( hipBindTexture((size_t)0, tex_eta_ap_ON, deta_ap_ON, nf_src * sizeof(float)) );
    gpuErrchk( hipBindTexture((size_t)0, tex_eta_ap_OFF, deta_ap_OFF, nf_src * sizeof(float)) );

    // Allocate and copy atmosphere arrays
    float *deta_atm;
    int neta_atm = _f_atm.num * _PWV_atm.num;
    
    gpuErrchk( hipMalloc((void**)&deta_atm, neta_atm * sizeof(float)) );
    gpuErrchk( hipMemcpy(deta_atm, eta_atm, neta_atm * sizeof(float), hipMemcpyHostToDevice) );
    delete[] eta_atm;

    // Allocate and copy instrument arrays
    float *dfilterbank;
    int nfilterbank = nf_src * instrument->nf_ch;
    gpuErrchk( hipMalloc((void**)&dfilterbank, nfilterbank * sizeof(float)) );
    
    gpuErrchk( hipMemcpy(dfilterbank, instrument->filterbank, nfilterbank * sizeof(float), hipMemcpyHostToDevice) );

    gpuErrchk( hipBindTexture((size_t)0, tex_filterbank, dfilterbank, nfilterbank * sizeof(float)) );
    
    timer.stop();

    //output->t_diag[0] = timer.get();

    gpuErrchk( hipMalloc((void**)&d_I_nu, source->nI_nu * sizeof(float)) );
    gpuErrchk( hipMemcpy(d_I_nu, source->I_nu, source->nI_nu * sizeof(float), hipMemcpyHostToDevice) );

    std::string datp;

    // Loop starts here
    printf("\033[92m");
    int idx_wrap = 0;
    int time_counter = 0;
    for(int idx=0; idx<nJobs; idx++) {
        if (idx_wrap == meta[0]) {
            idx_wrap = 0;
        }

        if (idx == (nJobs - 1)) {
            nTimesScreen = nTimesTotal - nTimesScreen*(nJobs-1);
        }
        time_counter += nTimesScreen;

        printf("*** Progress: %d / 100 ***\r", time_counter*100 / nTimesTotal);
        fflush(stdout);

        nffnt = instrument->nf_ch * nTimesScreen; // Number of elements in single-screen output.
        gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL(cnt), &nTimesScreen, sizeof(int)) );
        
        nBlocks1D = ceilf((float)nTimesScreen / NTHREADS1D / numSMs);
        blockSize1D = NTHREADS1D;
        gridSize1D = nBlocks1D*numSMs;
        nBlocks2Dx = ceilf((float)nTimesScreen / NTHREADS2DX / numSMs);
        nBlocks2Dy = ceilf((float)nf_src / NTHREADS2DY / numSMs);

        blockSize2D = dim3(NTHREADS2DX, NTHREADS2DY);
        gridSize2D = dim3(nBlocks2Dx * numSMs, nBlocks2Dy * numSMs);
    
        // Allocate output arrays
        gpuErrchk( hipMalloc((void**)&d_sigout, nffnt * sizeof(float)) );
        gpuErrchk( hipMalloc((void**)&d_nepout, nffnt * sizeof(float)) );
        gpuErrchk( hipMalloc((void**)&d_azout, nTimesScreen * sizeof(float)) );
        gpuErrchk( hipMalloc((void**)&d_elout, nTimesScreen * sizeof(float)) );
        gpuErrchk( hipMalloc((void**)&d_flagout, nTimesScreen * sizeof(int)) );

        // Allocate PWV screen now, delete CUDA allocation after first kernel call
        float *PWV_screen;
        float *dPWV_screen;
        
        int nPWV_screen = _x_atm.num * _y_atm.num;
        
        float *PWV_out;
        gpuErrchk( hipMalloc((void**)&PWV_out, nTimesScreen * sizeof(float)) );
        
        float pointing_center[2] = {0., 0.};
        float *dpointing_center;
        gpuErrchk( hipMalloc((void**)&dpointing_center, 2 * sizeof(float)) );
        gpuErrchk( hipMemcpy(dpointing_center, pointing_center, 2 * sizeof(float), hipMemcpyHostToDevice) );
        
        hiprandState *devStates;
        gpuErrchk( hipMalloc((void **)&devStates, nTimesScreen * sizeof(hiprandState)) );

        datp = std::to_string(idx_wrap) + ".datp";
        readAtmScreen<float, ArrSpec<float>>(&PWV_screen, &_x_atm, &_y_atm, str_path, datp);
        
        gpuErrchk( hipMalloc((void**)&dPWV_screen, nPWV_screen * sizeof(float)) );
        gpuErrchk( hipMemcpy(dPWV_screen, PWV_screen, nPWV_screen * sizeof(float), hipMemcpyHostToDevice) );
        

        get_chop_pwv_rng<<<gridSize1D, blockSize1D>>>(_Az_src, _El_src, _x_atm, _y_atm, dpointing_center, dPWV_screen, PWV_out, d_flagout, d_azout, d_elout, devStates);
       
        gpuErrchk( hipFree(dpointing_center) );
        gpuErrchk( hipFree(dPWV_screen) );

        // CALL TO MAIN SIMULATION KERNEL
        timer.start();
        
        // SINGLE POINTING, NO CHOP
        if(telescope->scantype == 0 && telescope->chop_mode == 0) {
            calcPowerNEP_1<<<gridSize2D, blockSize2D>>>(_f_spec, _f_atm, _PWV_atm, d_sigout, d_nepout, d_flagout,
                PWV_out, deta_atm, d_I_nu);
        }
        
        // SINGLE POINTING, STRICT ON-OFF
        else if(telescope->scantype == 0 && telescope->chop_mode == 1) {
            calcPowerNEP_2<<<gridSize2D, blockSize2D>>>(_f_spec, _f_atm, _PWV_atm, d_sigout, d_nepout, d_flagout,
                PWV_out, deta_atm, d_I_nu);
        }


        // SINGLE POINTING, ABBA CHOPPING
        else if(telescope->scantype == 0 && telescope->chop_mode == 2) {
            calcPowerNEP_3<<<gridSize2D, blockSize2D>>>(_f_spec, _f_atm, _PWV_atm, d_sigout, d_nepout, d_flagout,
                PWV_out, deta_atm, d_I_nu);
        }

        else {
            calcPowerNEP<<<gridSize2D, blockSize2D>>>(_f_spec, _f_atm, _PWV_atm, _Az_src, _El_src, d_sigout, d_nepout, d_azout, d_elout, d_flagout,
                    PWV_out, deta_atm, d_I_nu);
        }
        
        gpuErrchk( hipDeviceSynchronize() );

        gpuErrchk( hipFree(PWV_out) );
        
        calcPhotonNoise<<<gridSize1D, blockSize1D>>>(d_sigout, d_nepout, devStates);

        gpuErrchk( hipDeviceSynchronize() );
        timer.stop();
        
        gpuErrchk( hipFree(devStates) );
        gpuErrchk( hipFree(d_nepout) );

        //output->t_diag[1] = timer.get();
        

        //timer.start();
        // ALLOCATE STRINGS FOR WRITING OUTPUT
        std::string signame = std::to_string(idx) + "signal.out";
        std::string azname = std::to_string(idx) + "az.out";
        std::string elname = std::to_string(idx) + "el.out";
        std::string flagname = std::to_string(idx) + "flag.out";

        std::vector<float> sigout(nffnt);
        std::vector<float> azout(nTimesScreen);
        std::vector<float> elout(nTimesScreen);
        std::vector<int> flagout(nTimesScreen);

        gpuErrchk( hipMemcpy(sigout.data(), d_sigout, nffnt * sizeof(float), hipMemcpyDeviceToHost) );
        gpuErrchk( hipMemcpy(azout.data(), d_azout, nTimesScreen * sizeof(float), hipMemcpyDeviceToHost) );
        gpuErrchk( hipMemcpy(elout.data(), d_elout, nTimesScreen * sizeof(float), hipMemcpyDeviceToHost) );
        gpuErrchk( hipMemcpy(flagout.data(), d_flagout, nTimesScreen * sizeof(int), hipMemcpyDeviceToHost) );

        timer.start();

        write1DArray<float>(sigout, str_outpath, signame);
        write1DArray<float>(azout, str_outpath, azname);
        write1DArray<float>(elout, str_outpath, elname);
        write1DArray<int>(flagout, str_outpath, flagname);
        
        timer.stop();
        
        gpuErrchk( hipFree(d_sigout) );
        gpuErrchk( hipFree(d_azout) );
        gpuErrchk( hipFree(d_elout) );
        gpuErrchk( hipFree(d_flagout) );

        idx_wrap++;
    }
    gpuErrchk( hipDeviceReset() );
    timer.stop();
    output->t_diag[2] = timer.get();
    printf("\033[0m\n");
}

